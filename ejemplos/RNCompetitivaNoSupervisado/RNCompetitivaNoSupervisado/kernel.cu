
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

hipError_t addWithCuda(double *c, const int *a, const double *b, unsigned int sizen
	, unsigned int sizem);

__global__ void addKernel(double *c, const int *a, const double *b)
{
    int i = threadIdx.x;
	int j = 0;
	c[i] = 0;
	for (j = 0; j < 5; j++){
		
		c[i] += a[j] * b[i * 5 + j];
		printf("%d : ci %f, aj %d, b %f \n", i, c[i], a[j] , b[j]);
	}
	printf("final %d : %f \n",i,c[i]);
}

int main()
{
    const int n = 5;
	const int m = 5;
    const int a[n] = { 1,1,0,0,1 };
	const double b[n*m] = { 1, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 1 };
    double c[m] = { 0,0,0,0,0 };
	int t;

	for (t = 0; t < 25; t++){
		printf("%d : %f\n",t,b[t]);
	}

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, n, m);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	printf("lal");
    printf(" {%f,%f,%f,%f,%f}\n",
        c[0], c[1], c[2], c[3], c[4]);

	system("pause");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *c, const int *a, const double *b, unsigned int sizen, unsigned int sizem)
{
    int *dev_a = 0;
    double *dev_b = 0;
    double *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, sizem * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, sizen * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, sizen*sizem * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, sizen * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, sizen*sizem * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, sizem>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, sizem * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
