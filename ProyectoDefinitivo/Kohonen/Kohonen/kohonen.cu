#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "kohonen.h"
#include "hip/device_functions.h"

#include <stdio.h>
#include <math.h>

__shared__ float map_shared[kohonen::mapSize * kohonen::dimension];
__shared__ float input_shared[kohonen::inputSize*kohonen::numInput];
__shared__ float weight_shared[kohonen::inputSize*kohonen::mapSize];


__global__ void startData(int inputSize, float maxInputX, float minInputX, float maxInputY, float minInputY, float *dev_input,  float *dev_map)
{
	int i = threadIdx.x;
	if (i < kohonen::numInput){
		input_shared[i*2] = (dev_input[i*2]-(minInputX+maxInputX)/2)/(maxInputX-minInputX);
		input_shared[i * 2 + 1] = (dev_input[i * 2 + 1] - (minInputY + maxInputY) / 2) / (maxInputY - minInputY);
	}
	int j;
	for (j = 0; j < 2; j++){
		map_shared[i * 2 +j] = dev_map[i * 2+j];
		map_shared[i * 2 + j] = dev_map[i * 2 + j];
	}
	for (j = 0; j < inputSize; j++){
		weight_shared[i*inputSize + j] = 0.5;
	}
    
}

__global__ void learnApuntes(int mapSize,int inputSize){
	int minMapLeft1, minMapRight1, minMapLeft2, minMapRight2;
	float eta = 0.1f;
	int i = threadIdx.x;
	float hI = 0.0f;
	int minMap = 0;
	int nodo;
	float hR;
	int epoch;
	for (epoch = 0; epoch < 1000; epoch++){
		hR = 0.0f;
		//sacar la neurona ganadora y sus vecinos
		for (nodo= 0; nodo < mapSize; nodo++){
			hR = hR + weight_shared[nodo*inputSize]/2 * input_shared[i*inputSize] + weight_shared[nodo*inputSize + 1]/2 * input_shared[i*inputSize + 1];
			if (hR < hI){
				hI = hR;
				minMap = nodo;
			}
		}
		minMapRight1 = (minMap+1) % mapSize;
		minMapLeft1 = (minMap - 1) % mapSize;
		minMapRight2 = (minMap + 2) % mapSize;
		minMapLeft2 = (minMap -2) % mapSize;
		
		weight_shared[minMap*inputSize] = weight_shared[minMap*inputSize] + eta*(input_shared[i*inputSize] - weight_shared[minMap*inputSize]);
		weight_shared[minMap*inputSize + 1] = weight_shared[minMap*inputSize+1] + eta*(input_shared[i*inputSize+1] - weight_shared[minMap*inputSize+1]);
		__syncthreads();

		weight_shared[minMapLeft1*inputSize] = weight_shared[minMapLeft1*inputSize] + eta*0.5*(input_shared[i*inputSize] - weight_shared[minMapLeft1*inputSize]);
		weight_shared[minMapLeft1*inputSize + 1] = weight_shared[minMapLeft1*inputSize + 1] + eta*0.5*(input_shared[i*inputSize + 1] - weight_shared[minMapLeft1*inputSize + 1]);
		__syncthreads();

		weight_shared[minMapLeft2*inputSize]=weight_shared[minMapLeft2*inputSize] + eta*0.25*(input_shared[i*inputSize] - weight_shared[minMapLeft2*inputSize]);
		weight_shared[minMapLeft1*inputSize + 1] = weight_shared[minMapLeft2*inputSize + 1] + eta*0.25*(input_shared[i*inputSize + 1] - weight_shared[minMapLeft2*inputSize + 1]);
		__syncthreads();

		weight_shared[minMapRight1*inputSize] = weight_shared[minMapRight1*inputSize] + eta*0.5*(input_shared[i*inputSize] - weight_shared[minMapRight1*inputSize]);
		weight_shared[minMapLeft1*inputSize + 1] = weight_shared[minMapRight1*inputSize + 1] + eta*0.5*(input_shared[i*inputSize + 1] - weight_shared[minMapRight1*inputSize + 1]);
		__syncthreads();

		weight_shared[minMapRight2*inputSize] = weight_shared[minMapRight2*inputSize] + eta*0.25*(input_shared[i*inputSize] - weight_shared[minMapRight2*inputSize]);
		weight_shared[minMapLeft1*inputSize + 1] = weight_shared[minMapRight2*inputSize + 1] + eta*0.25*(input_shared[i*inputSize + 1] - weight_shared[minMapRight2*inputSize + 1]);
	}
}

__global__ void stopData(int inputSize, float *dev_map, float *dev_weight)
{
	int i = threadIdx.x;
	int j;
	for (j = 0; j < inputSize; j++){
		dev_weight[i*inputSize +j]=weight_shared[i*inputSize + j];
	}
	//for (j = 0; j < 2; j++){
	//	dev_map[i * 2 + j] = map_shared[i * 2 + j];
	//}
}

// Helper function for using CUDA to add vectors in parallel.
void kohonen::train(int inputSize, int mapSize, int numInput, float *input, float *map, float *weight, float maxInputX, float minInputX, float maxInputY, float minInputY)
{
	float *dev_input = 0;
	float *dev_map = 0;
	float *dev_weight = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_weight, inputSize*mapSize* sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_input, inputSize*numInput* sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_map, mapSize * dimension * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_weight, weight, inputSize*mapSize * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_input, input, inputSize*numInput * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_map, map, mapSize * 2 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
	startData << <1, mapSize >> >(inputSize, maxInputX, minInputX, maxInputY, minInputY, dev_input, dev_map);

	learnApuntes << <1, numInput >> >(mapSize,inputSize);
	
	stopData << <1, mapSize >> >(inputSize, dev_map,dev_weight);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(weight, dev_weight, mapSize * inputSize * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(map, dev_map, mapSize * 2 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
    hipFree(dev_input);
	hipFree(dev_map);
	hipFree(dev_weight);
    
    //return cudaStatus;
}
