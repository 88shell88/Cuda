#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "hip/hip_runtime.h"
#include ""
#include "kohonen.h"
#include "hip/device_functions.h"

__global__ void learnFirstIteration(int mapSize, int inputSize, int numInput, float maxInputX, float minInputX, float maxInputY, float minInputY, float *dev_input, float *dev_map){
	extern __shared__ float shared[];
	float *map_shared = shared;
	float *input_shared = (float*)&map_shared[mapSize * 2];
	float *hits_shared = (float*)&input_shared[numInput * 2];
	
	float eta = 0.1f;
	int i = threadIdx.x;
	float hI;
	int minMap = 0;
	int minNodo = 0;
	int nodo;
	float hR;
	int epoch;
	//start data
	
	
	if (i < kohonen::numInput){
		input_shared[i * 2] = dev_input[i * 2];
		input_shared[i * 2 + 1] = dev_input[i * 2 + 1] ;
		hits_shared[i] = 0;
	}

	map_shared[i * 2] = dev_map[i * 2];
	map_shared[i * 2 + 1] = dev_map[i * 2 + 1];
	__syncthreads();

	for (epoch = 0; epoch < 50; epoch++){
		hR = 0.0f;
		//sacar la neurona ganadora y sus vecinos
		hI = sqrt(pow((input_shared[0 * inputSize] - map_shared[i*inputSize]), 2) + pow((input_shared[0 * inputSize + 1] - map_shared[i*inputSize + 1]), 2));
		for (nodo = 1; nodo < numInput; nodo++){
			hR = sqrt(pow((input_shared[nodo*inputSize] - map_shared[i*inputSize]), 2) + pow((input_shared[nodo*inputSize + 1] - map_shared[i*inputSize + 1]), 2)) *((hits_shared[nodo]+1) / (epoch + 1));
			if (hR < hI){

				hI = hR;
				minNodo = nodo;

			}

		}
		hits_shared[minNodo] = hits_shared[minNodo] + 1;
		minMap = i;
		
		map_shared[minMap*inputSize] = map_shared[minMap*inputSize] + eta*(input_shared[minNodo*inputSize] - map_shared[minMap*inputSize]);
		map_shared[minMap*inputSize + 1] = map_shared[minMap*inputSize + 1] + eta*(input_shared[minNodo*inputSize + 1] - map_shared[minMap*inputSize + 1]);

	}
	//stop data
	dev_map[i*inputSize] = map_shared[i*inputSize];
	dev_map[i*inputSize + 1] = map_shared[i*inputSize + 1];
}



__global__ void learnSecondIteration(int mapSize, int inputSize, int numInput, float maxInputX, float minInputX, float maxInputY, float minInputY, float *dev_input, float *dev_map){
	extern __shared__ float shared[];
	float *map_shared = shared;
	float *input_shared = (float*)&map_shared[mapSize * 2];

	int minMapLeft1, minMapRight1, minMapLeft2, minMapRight2;
	float eta = 0.1f;
	int i = threadIdx.x;
	float hI;
	int minMap = 0;
	int minNodo = 0;
	int nodo;
	float hR;
	int epoch;

	//start data
	input_shared[i * 2] = dev_input[i * 2];
	input_shared[i * 2 + 1] = dev_input[i * 2 + 1];


	map_shared[i * 6] = dev_map[i * 6];
	map_shared[i * 6 + 1] = dev_map[i * 6 + 1];
	map_shared[i * 6 + 2] = dev_map[i * 6 + 2];
	map_shared[i * 6 + 3] = dev_map[i * 6 + 3];
	map_shared[i * 6 + 4] = dev_map[i * 6 + 4];
	map_shared[i * 6 + 5] = dev_map[i * 6 + 5];


	__syncthreads();

	
	//Iteraciones desde 0 hasta X
	
	for (epoch = 0; epoch < 500; epoch++){
		// hR es la distancia a comprobar, empieza a 0
		hR = 0.0f;

		//sacar la neurona ganadora y sus vecinos
		//hI es el minimo local
		hI = sqrt(pow((input_shared[0] - map_shared[i*inputSize]), 2) + pow((input_shared[1] - map_shared[i*inputSize + 1]), 2));
		//Por cada nodo que no sea el primero
		for (nodo = 1; nodo < mapSize; nodo++){
			//calcula la distancia de ese nodo

			hR = sqrt(pow((input_shared[i*inputSize] - map_shared[nodo*inputSize]), 2) + pow((input_shared[i*inputSize + 1] - map_shared[nodo*inputSize + 1]), 2));
			if ((hR < hI)){
				hI = hR;
				minMap = nodo;

			}

		}

		minNodo = i;
		//necesito almacenar el indice de los nodos siguientes
		minMapRight1 = minMap + 1;
		minMapRight2 = minMap + 2;
		if (minMapRight1 == mapSize) {
			minMapRight1 = 0;
			minMapRight2 = 1;
		}
		if (minMapRight2 == mapSize) minMapRight2 = 0;

		//y los dos nodos anteriores
		minMapLeft1 = minMap - 1;
		minMapLeft2 = minMap - 2;
		if (minMapLeft1 == -1)  {
			
			minMapLeft1 = mapSize - 1;
			minMapLeft2 = mapSize - 2;
		}
		if (minMapLeft2 == -1) minMapLeft2 = mapSize - 1;
		eta = eta - eta / 100;
		if (epoch >= 100) eta = 0.5f;
		map_shared[minMap*inputSize] = map_shared[minMap*inputSize] + eta*(input_shared[minNodo*inputSize] - map_shared[minMap*inputSize]);
		map_shared[minMap*inputSize + 1] = map_shared[minMap*inputSize + 1] + eta*(input_shared[minNodo*inputSize + 1] - map_shared[minMap*inputSize + 1]);

		if (epoch < 100){
			map_shared[minMapLeft1*inputSize] = map_shared[minMapLeft1*inputSize] + eta*0.5*(input_shared[minNodo*inputSize] - map_shared[minMapLeft1*inputSize]);
			map_shared[minMapLeft1*inputSize + 1] = map_shared[minMapLeft1*inputSize + 1] + eta*0.5*(input_shared[minNodo*inputSize + 1] - map_shared[minMapLeft1*inputSize + 1]);

			map_shared[minMapLeft2*inputSize] = map_shared[minMapLeft2*inputSize] + eta*0.25*(input_shared[minNodo*inputSize] - map_shared[minMapLeft2*inputSize]);
			map_shared[minMapLeft2*inputSize + 1] = map_shared[minMapLeft2*inputSize + 1] + eta*0.25*(input_shared[minNodo*inputSize + 1] - map_shared[minMapLeft2*inputSize + 1]);

			map_shared[minMapRight1*inputSize] = map_shared[minMapRight1*inputSize] + eta*0.5*(input_shared[minNodo*inputSize] - map_shared[minMapRight1*inputSize]);
			map_shared[minMapRight1*inputSize + 1] = map_shared[minMapRight1*inputSize + 1] + eta*0.5*(input_shared[minNodo*inputSize + 1] - map_shared[minMapRight1*inputSize + 1]);

			map_shared[minMapRight2*inputSize] = map_shared[minMapRight2*inputSize] + eta*0.25*(input_shared[minNodo*inputSize] - map_shared[minMapRight2*inputSize]);
			map_shared[minMapRight2*inputSize + 1] = map_shared[minMapRight2*inputSize + 1] + eta*0.25*(input_shared[minNodo*inputSize + 1] - map_shared[minMapRight2*inputSize + 1]);
		}
	}
	__syncthreads();
	//stop data
	dev_map[i * 6] = map_shared[i * 6];
	dev_map[i * 6 + 1] = map_shared[i * 6 + 1];
	dev_map[i * 6 + 2] = map_shared[i * 6 + 2];
	dev_map[i * 6 + 3] = map_shared[i * 6 + 3];
	dev_map[i * 6 + 4] = map_shared[i * 6 + 4];
	dev_map[i * 6 + 5] = map_shared[i * 6 + 5];
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t kohonen::train(int inputSize, int mapSize, int numInput, float *input, float *map, float maxInputX, float minInputX, float maxInputY, float minInputY)
{
	
	float *dev_input = 0;
	float *dev_map = 0;

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_input, inputSize*numInput* sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_map, mapSize * dimension * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	
    // Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input, input, inputSize*numInput * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_map, map, mapSize * 2 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

    // Launch a kernel on the GPU with one thread for each element.
	learnFirstIteration << <1, mapSize , sizeof(float)*(mapSize*2+numInput*2 + numInput)>> >(mapSize, inputSize, numInput, maxInputX, minInputX, maxInputY, minInputY, dev_input, dev_map);
	learnSecondIteration << <1, numInput, sizeof(float)*(mapSize * 2 + numInput * 2) >> >(mapSize, inputSize, numInput, maxInputX, minInputX, maxInputY, minInputY, dev_input, dev_map);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(map, dev_map, mapSize * 2 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
    hipFree(dev_input);
	hipFree(dev_map);
    
    return cudaStatus;
}
